// PABLO ANDRES COUTINHO BURGOS
// AUGUSTO ESTUARDO ALONSO ASCENCIO

#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <utility>
#include <stdio.h>
#include <string>
#include <cmath>
#include <math.h>

using namespace std;
__global__
void getTotal(int n, float *array, float * totalL)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if (index < n) totalL[0] += (array[index]);

}
__global__

void getVariance(int n, float *array, float * mean, float * vairance)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if (index < n) vairance[0] += (pow(array[index] - mean[0], 2));
}

// vector<string> explode(string const & s, char delim)
// {
//     vector<string> result;
//     istringstream iss(s);

//     for (string token; getline(iss, token, delim); )
//     {
//         result.push_back(move(token));
//     }

//     return result;
// }

int main(void)
{
	int N = 100000;
	float *hoursArray, *cudaHours;
	float *temperaturesArray, *cudaTemperatures;
	hoursArray = (float*)malloc(N*sizeof(float));
	temperaturesArray = (float*)malloc(N*sizeof(float));
	hipMalloc(&cudaHours, N*sizeof(float));
	hipMalloc(&cudaTemperatures, N*sizeof(float));

	const char *ts[4] = {"temps1.txt", "temps2.txt", "temps3.txt", "temps4.txt"};
	const char *hs[4] = {"hours1.txt", "hours2.txt", "hours3.txt", "hours4.txt"};
	    vector<string> hoursArrayS;
	    vector<string> tempsArrayS;
	for (int i = 1; i < 5; ++i)
	{

		int offset = (i - 1) * 25000;
		ifstream hoursFile;
		ifstream tempsFile;
		hoursFile.open(hs[i-1]);
		tempsFile.open(ts[i-1]);
		string hours = "";
		string temps = "";
		string line ="";
		string lineT ="";
		while(getline(hoursFile,line))
	    {
	      hours += line;
	    }
		while(getline(tempsFile,lineT))
	    {
	      temps += lineT;
	    }


	    for (int i = 0; i < 3000; i+=3)
	    {
	    	string s = hours.substr(i, 2);

	    	hoursArrayS.push_back(hours.substr(i, 2));
	    }

	    for (int i = 0; i < 3000; i+=3)
	    {
	    	tempsArrayS.push_back(temps.substr(i, 2));
	    }
	    for (int j= 0; j< hoursArrayS.size(); ++j)
	    {
	    	char* pEnd;
	    	hoursArray[j + offset] = strtof(hoursArrayS[j].c_str(),&pEnd);
	    	temperaturesArray[j + offset] = strtof(tempsArrayS[j].c_str(),&pEnd);
	    	// printf("%i\n", (j + offset));
	    	// printf("%f\n", hoursArray[j + offset]);
	    }
		hoursFile.close();
		tempsFile.close();
			/* code */
	}
	hipMemcpy(cudaHours, hoursArray, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaTemperatures, temperaturesArray, N*sizeof(float), hipMemcpyHostToDevice);
	// Statistics calcs
	float *totalTemps, *totalHours, *vairanceHours, *vairanceTemperature, *meanHours, *meanTemperature, standardDeviationTemperature;
	float *totalTempsCuda, *totalHoursCuda, *vairanceHoursCuda, *vairanceTemperatureCuda, *meanHoursCuda, *meanTemperatureCuda;
	totalTemps = (float*)malloc(1*sizeof(float));
	totalHours = (float*)malloc(1*sizeof(float));
	vairanceHours = (float*)malloc(1*sizeof(float));
	vairanceTemperature = (float*)malloc(1*sizeof(float));
	meanHours = (float*)malloc(1*sizeof(float));
	meanTemperature = (float*)malloc(1*sizeof(float));
	hipMalloc(&totalTempsCuda, 1*sizeof(float));
	hipMalloc(&totalHoursCuda, 1*sizeof(float));
	hipMalloc(&vairanceHoursCuda, 1*sizeof(float));
	hipMalloc(&vairanceTemperatureCuda, 1*sizeof(float));
	hipMalloc(&meanHoursCuda, 1*sizeof(float));
	hipMalloc(&meanTemperatureCuda, 1*sizeof(float));
	// totalHours = 0.0f;
	// totalTemps = 0.0f;
	// vairanceHours = 0.0f;
	// vairanceTemperature = 0.0f;
	getTotal<<<hoursArrayS.size()/4, 4>>>(hoursArrayS.size(), cudaHours, totalHoursCuda);
	getTotal<<<hoursArrayS.size()/4, 4>>>(hoursArrayS.size(), cudaTemperatures, totalTempsCuda);

	hipDeviceSynchronize();
	hipMemcpy(totalHours, totalHoursCuda, 1*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(totalTemps, totalTempsCuda, 1*sizeof(float), hipMemcpyHostToDevice);
	printf("%f\n", totalHours[0]);
	meanTemperature[0] = totalTemps[0]/N;
	meanHours[0] = totalHours[0]/N;
	hipMemcpy(meanTemperatureCuda, meanTemperature, 1*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(meanHoursCuda, meanHours, 1*sizeof(float), hipMemcpyHostToDevice);
	getVariance<<<hoursArrayS.size()/4, 4>>>(hoursArrayS.size(), cudaHours, meanHoursCuda, vairanceHoursCuda);
	getVariance<<<hoursArrayS.size()/4, 4>>>(hoursArrayS.size(), cudaTemperatures, meanTemperatureCuda, vairanceTemperatureCuda);

	hipDeviceSynchronize();
	hipMemcpy(vairanceHoursCuda, vairanceHours, 1*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vairanceTemperatureCuda, vairanceTemperature, 1*sizeof(float), hipMemcpyHostToDevice);
	vairanceHours[0] = vairanceHours[0]/hoursArrayS.size();
	vairanceTemperature[0] = vairanceTemperature[0]/hoursArrayS.size();
	standardDeviationTemperature = sqrt(vairanceTemperature[0]);
	printf("La desviacion estandar de la temperatura entre las 13:00 - 16:00 fue de %f\n", standardDeviationTemperature);
	printf("La varianza de la temperatura entre las 13:00 - 16:00 fue de %f\n", vairanceTemperature[0]);
	printf("La media de la temperatura entre las 13:00 - 16:00 fue de %f\n", meanTemperature[0]);
    // for (int j= 0; j< N; ++j)
    // {
    // 	printf("%f\n", hoursArray[j]);
    // 	printf("%f\n", temperaturesArray[j]);
    // }
}